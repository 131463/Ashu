#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------
//
// This source file is part of the VAMPIRE open source package under the
// GNU GPL (version 2) licence (see licence file for details).
//
// (c) Andrea Meo 2022. All rights reserved.
//
//------------------------------------------------------------------------------

// C++ standard library headers

// Vampire headers
#include "cuda.hpp"
#include "errors.hpp"
#include "hamr.hpp"
#include "random.hpp"
#include "sim.hpp"
#include "vio.hpp"

// Local cuda headers
#include "cuda_utils.hpp"
#include "data.hpp"
#include "internal.hpp"
#include "thermal_fields.hpp"
#include "typedefs.hpp"

// Conditional compilation of all cuda code
#ifdef CUDA
// namespace aliasing for brevity
namespace cu = vcuda::internal;

// vampire cuda namespace
namespace vcuda{

	// module internal namespace
	namespace internal{
	
		// Function to calculate temperature of atom depending on Gaussian profile
		__device__ cu_real_t calculate_gaussian_profile(
			const int atom, const cu_real_t cx, const cu_real_t cy,
			const cu_real_t px, const cu_real_t py, 
			const cu_real_t Tmin, const cu_real_t DeltaT,
			const cu_real_t laser_sigma_x2, const cu_real_t laser_sigma_y2
			)
		{

			const cu_real_t cx2 = (cx-px)*(cx-px);
			const cu_real_t cy2 = (cy-py)*(cy-py);
			const cu_real_t denx = 2.0 * laser_sigma_x2;
			const cu_real_t one_over_denx = 1.0/denx;
			const cu_real_t deny = 2.0 * laser_sigma_y2;
			const cu_real_t one_over_deny = 1.0/deny;

			#ifdef CUDA_DP
				cu_real_t exp_x =  exp(-cx2*one_over_denx); 
				cu_real_t exp_y =  exp(-cy2*one_over_deny); 
			#else
				cu_real_t exp_x =  __expf(-cx2*one_over_denx); 
				cu_real_t exp_y =  __expf(-cy2*one_over_deny); 
			#endif
			// if(i<5){printf("  %d %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf\n",
			// 						i, cx, cy, px, py, DeltaT, denx, deny, one_over_denx, one_over_deny, exp_x, exp_y); }

			cu_real_t temperature = Tmin + DeltaT * exp_x * exp_y;

			return temperature;
		} // end calculate_gaussian_profile


		// Kernel to apply thermal field with local temperature
		__global__ void apply_local_temperature_kernel(
			 	cu_real_t * atoms_coord_x, cu_real_t * atoms_coord_y,
				cu_real_t * x_field_array, cu_real_t * y_field_array, cu_real_t * z_field_array,
				const cu_real_t Tmin, const cu_real_t Tmax,
				const cu_real_t laser_sigma_x2, const cu_real_t laser_sigma_y2,
				const cu_real_t px, const cu_real_t py,	 
				hiprandState * rand_states,
				cu::material_parameters_t * material_params,
				int * material,
				const int n_atoms
				)
		{
			int tid = blockIdx.x * blockDim.x + threadIdx.x;
			for (int i = tid; 
				i < n_atoms; 
				i += blockDim.x * gridDim.x)
			{

				// Get material of atom i
				int mid = material[i];
				// Load parameters from memory
				cu::material_parameters_t mat = material_params[mid];
				// Load the hiprand state into local memory
				hiprandState local_state = rand_states[tid];

				// Define temporary variables for field
				cu_real_t field_x = 0.0;
				cu_real_t field_y = 0.0;
				cu_real_t field_z = 0.0;

				// Assign tempeerature to atoms according to Gaussian profile
				const cu_real_t DeltaT = Tmax - Tmin;
				cu_real_t temp = calculate_gaussian_profile(i, atoms_coord_x[i], atoms_coord_y[i], px, py,
																Tmin, DeltaT, laser_sigma_x2, laser_sigma_y2);
				
				// material dependent temperature rescaling
				const cu_real_t alpha = mat.temperature_rescaling_alpha;
				const cu_real_t Tc    = mat.temperature_rescaling_Tc;
				const cu_real_t sigma = mat.H_th_sigma;

      		// thermal field calculation

      		#ifdef CUDA_DP
      		   double resc_temp = (temp < Tc) ? Tc * pow(temp / Tc, alpha) : temp;
      		   double rsigma = sigma*sqrt(resc_temp);
      		#else
      		   float resc_temp = (temp < Tc) ? Tc * __powf(temp / Tc, alpha) : temp;
      		   float rsigma = sigma*sqrtf(resc_temp);
      		#endif

      		#ifdef CUDA_DP
      		   field_x = rsigma * hiprand_normal_double (&local_state);
      		   field_y = rsigma * hiprand_normal_double (&local_state);
      		   field_z = rsigma * hiprand_normal_double (&local_state);
      		#else
      		   field_x = rsigma * hiprand_normal(&local_state);
      		   field_y = rsigma * hiprand_normal(&local_state);
      		   field_z = rsigma * hiprand_normal(&local_state);
      		#endif

      		x_field_array[i] = field_x;
      		y_field_array[i] = field_y;
      		z_field_array[i] = field_z;

      		// Write local hiprand state back to global memory
      		rand_states[tid] = local_state;
			}

			return;
		} // end apply_local_temperature_kernel



		// Kernel to apply thermal field with local temperature
		__global__ void apply_local_external_field_kernel(
			 	cu_real_t * atoms_coord_x, cu_real_t * atoms_coord_y,
				cu_real_t * x_field_array, cu_real_t * y_field_array, cu_real_t * z_field_array,
				const cu_real_t Hx_app, const cu_real_t Hy_app, const cu_real_t Hz_app, 
				const cu_real_t head_position_x, const cu_real_t head_position_y,
				const cu_real_t H_bounds_x, const cu_real_t H_bounds_y, 
				const cu_real_t NPS,
				const int n_atoms
				)
		{
			for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
				i < n_atoms; 
				i += blockDim.x * gridDim.x)
			{

				const cu_real_t cx = atoms_coord_x[i]; 
				const cu_real_t cy = atoms_coord_y[i]; 
				const cu_real_t Hloc_min_x = head_position_x - H_bounds_x - NPS;
				const cu_real_t Hloc_max_x = head_position_x + H_bounds_x - NPS;
				const cu_real_t Hloc_min_y = head_position_y - H_bounds_y;
				const cu_real_t Hloc_max_y = head_position_y + H_bounds_y;

				// If atoms within field box, add contribution from external field
				if((cx >= Hloc_min_x) && (cx <= Hloc_max_x) && (cy >= Hloc_min_y) && (cy <= Hloc_max_y)){
					x_field_array[i] += Hx_app;
					y_field_array[i] += Hy_app;
					z_field_array[i] += Hz_app;
				}

			}

			return;
		} // end apply_local_external_field_kernel


		// Function to calculate update hamr field
		void update_hamr_field(
			)
		{
			// Check that hamr field calculation has been called
			if(err::check==true){ std::cout << "calculate_hamr_fields has been called" << std::endl;}

			// copy simulation variables to temporary constants
			const cu_real_t global_temperature = sim::temperature;
			const int num_atoms = ::atoms::num_atoms;

			// // Initialise to zero hamr fields
			// hipMemset(cu::d_x_hamr_field, 0, num_atoms * sizeof(cu_real_t));
			// hipMemset(cu::d_y_hamr_field, 0, num_atoms * sizeof(cu_real_t));
			// hipMemset(cu::d_z_hamr_field, 0, num_atoms * sizeof(cu_real_t));

			check_cuda_errors (__FILE__, __LINE__);

			if(::hamr::head_laser_on){

				check_cuda_errors (__FILE__, __LINE__);

				const cu_real_t Tmin = sim::Tmin;
				const cu_real_t Tmax = sim::Tmax;
				const cu_real_t Hx_app = sim::H_vec[0]*sim::H_applied;
				const cu_real_t Hy_app = sim::H_vec[1]*sim::H_applied;
				const cu_real_t Hz_app = sim::H_vec[2]*sim::H_applied;
				// Determine constants
				const cu_real_t H_bounds_x = cu::hamr::d_H_bounds_x;
				const cu_real_t H_bounds_y = cu::hamr::d_H_bounds_y;
				const cu_real_t NPS = cu::hamr::d_NPS;
				const cu_real_t laser_sigma_x2 = cu::hamr::d_laser_sigma_x * cu::hamr::d_laser_sigma_x;
				const cu_real_t laser_sigma_y2 = cu::hamr::d_laser_sigma_y * cu::hamr::d_laser_sigma_y;
				// Update head position - updated in src/hamr/hamr_continuous.cpp
				cu::hamr::d_head_position_x = ::hamr::get_head_position_x();
				cu::hamr::d_head_position_y = ::hamr::get_head_position_y();
				const cu_real_t px = cu::hamr::d_head_position_x;
				const cu_real_t py = cu::hamr::d_head_position_y;

				// Apply thermal field
				apply_local_temperature_kernel <<< cu::grid_size, cu::block_size >>> (
					cu::atoms::d_x_coord, cu::atoms::d_y_coord,
					cu::d_x_hamr_field, cu::d_y_hamr_field, cu::d_z_hamr_field,
					Tmin, Tmax,
					laser_sigma_x2, laser_sigma_y2,
					px, py,	 
					cu::d_rand_state,
					cu::mp::d_material_params,
					cu::atoms::d_materials,
					num_atoms);

				check_cuda_errors (__FILE__, __LINE__);

				// Apply external field 
				apply_local_external_field_kernel <<< cu::grid_size, cu::block_size >>> (
					cu::atoms::d_x_coord, cu::atoms::d_y_coord,
					cu::d_x_hamr_field, cu::d_y_hamr_field, cu::d_z_hamr_field,
					Hx_app, Hy_app, Hz_app,
					px, py,
					H_bounds_x, H_bounds_y, 
					NPS,
					num_atoms);

				check_cuda_errors (__FILE__, __LINE__);
			}
			// Apply global temperature if laser is off
			else{

				check_cuda_errors (__FILE__, __LINE__);

				cu::apply_global_temperature_kernel <<< cu::grid_size, cu::block_size >>> (
					cu::d_x_hamr_field, cu::d_y_hamr_field, cu::d_z_hamr_field,
					global_temperature,
					cu::d_rand_state,
					cu::mp::d_material_params,
					cu::atoms::d_materials,
					num_atoms); 

				check_cuda_errors (__FILE__, __LINE__);
			}

			return;
		} // end update_hamr_field


	} // end namespace internal

} // end namespace vcuda


#endif